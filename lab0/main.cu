#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

const int W = 40;
const int H = 12;

__global__ void DrawStandard(char *frame) 
{
	// TODO: draw more complex things here
	// Do not just submit the original file provided by the TA!
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (y < H && x < W) {
		char c;
		if (x == W - 1) 
			c = (y == H - 1) ? '\0' : '\n';
		else if (y == 0 || y == H - 1 || x == 0 || x == W - 2) 
			c = ':';//BOX
		else if (y >= 5 && y <= 10 && x <= 21 && x >= 8 + (10 - y) * 2)
			c = '#';//Stair
		else if (y >= 5 && y <= 10 && x == 33)
			c = (y == 10) ? '#' : '|';//Flag
		else if (y == 5 && x == 32)
			c = '<';//Flag
		else 
			c = ' ';

		frame[y*W + x] = c;
	}
}

int main(int argc, char **argv)
{
	MemoryBuffer<char> frame(W*H);
	auto frame_smem = frame.CreateSync(W*H);
	CHECK;

	DrawStandard << <dim3((W - 1) / 16 + 1, (H - 1) / 12 + 1), dim3(16, 12) >> >(frame_smem.get_gpu_wo());
	CHECK;

	puts(frame_smem.get_cpu_ro());
	CHECK;
	
	return 0;
}